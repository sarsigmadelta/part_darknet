#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void vector_add(float *a, float *b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] += b[i];
}

__global__ void pixel_remove(image* im){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int c = 0;
    for(c=0; c<im->channels; ++c){
        int index = c * im->height * im->width + row * im->width + col;
        im->data[index] = 0.1;
    }
}

void vector_add_called(float *a, float *b){
    dim3 blockSize(1, 1);
    dim3 gridSize(1, 1);
    vector_add<<<blockSize, gridSize>>>(a, b);
}

void image_smooth_test(image im){
    size_t nBytes = im.height * im.width * im.channels * sizeof(float);
    image *im_d ;
    hipMallocManaged((void**)&im_d, sizeof(image));
    im_d->width = im.width;
    im_d->height = im.height;
    im_d->channels = im.channels;
    
    hipMallocManaged((void**)&im_d->data, nBytes);
    hipMemcpy(im_d->data, im.data, nBytes, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(16, 16);
    
    pixel_remove<<<gridSize,  blockSize>>>(im_d);

    hipMemcpy(im.data, im_d->data, nBytes, hipMemcpyDeviceToHost);
    
}
