#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void vector_add(float *a, float *b){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] += b[i];
}

__global__ void pixel_remove(image* im){
    int idx_x = threadIdx.x + blockDim.x * blockDim.x;
    int idx_y = threadIdx.y + blockDim.y * blockDim.y;
    int index = idx_y * im->width + idx_x;
    im->data[index] *= 0.0;
    //printf("im->data[index] %f\n", im->data[index]);
}

void vector_add_called(float *a, float *b){
    dim3 blockSize(1, 1);
    dim3 gridSize(1, 1);
    vector_add<<<blockSize, gridSize>>>(a, b);
}

void image_smooth_test(image im){
    size_t nBytes = im.height * im.width * im.channels * sizeof(float);
    image *im_d ;
    hipMallocManaged((void**)&im_d, sizeof(image));
    hipMallocManaged((void**)&im_d->data, nBytes);
    hipMemcpy(im_d->data, im.data, nBytes, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(16, 16);
    
    pixel_remove<<<gridSize,  blockSize>>>(im_d);

    hipMemcpy(im.data, im_d->data, nBytes, hipMemcpyDeviceToHost);
    
}
