#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void vector_add(float *a, float *b){
    
}

void vector_add_called(float *a, float *b){
    dim3 blockSize(32, 32);
    dim3 gridSize(1, 1);
    vector_add<<<blockSize, gridSize>>>(a, b);
}