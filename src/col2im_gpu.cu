#include "hip/hip_runtime.h"
#include "col2im.h"

const int blockSize = 512;


__global__ void col2im_gpu_kernel(float *data_col, int channels, int height, int width,
                            int ksize, int stride, int pad, 
                            int height_col, int width_col, float *im){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int w_im = index % width;
    int h_im = index / width % height;
    int c_im = index / width / height;
    
    int w_col_start = ( w_im < ksize) ? 0 : (w_im - ksize) / stride + 1;
    int w_col_end = min(w_im / stride + 1, width_col);
    int h_col_start = (h_im < ksize ) ? 0 : (h_im - ksize) / stride + 1;
    int h_col_end = min(h_im / stride + 1, height_col);

    int h_col, w_col;
    for(h_col=h_col_start; h_col<h_col_end; ++h_col){
        for(w_col=w_col_start; w_col<w_col_end; ++w_col){
            int w_k = h_im - h_col * stride;
            int h_k = w_im - w_col * stride;
            int index_col = (c_im * ksize * ksize + h_k * ksize + w_k) * height_col * width_col + h_col * width_col + w_col;
            im[index] += data_col[index_col];
        }
    }
}


void col2im_gpu(float *data_col, 
            int channels, int height, int width,
            int ksize, int stride, int pad, float *im){
    int height_col = (height - ksize + 2*pad)/stride + 1;
    int width_col = (width - ksize + 2*pad)/stride + 1;
    int num_threads = height * width * channels;
    col2im_gpu_kernel<<<(num_threads + blockSize -1 )/blockSize, blockSize>>>(data_col, channels, height, width,
                                                ksize, stride, pad, height_col, width_col, im);
}