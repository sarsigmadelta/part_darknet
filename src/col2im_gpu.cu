#include "hip/hip_runtime.h"
#include "col2im.h"

const int blockSize = 512;


__global__ void col2im_gpu_kernel(float *data_col, int channels, int height, int width,
                            int ksize, int stride, int pad, 
                            int height_col, int width_col, float *im){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int 
}


void col2im_gpu(float *data_col, 
            int channels, int height, int width,
            int ksize, int stride, int pad, float *im){
    int height_col = (height - ksize + 2*pad)/stride + 1;
    int width_col = (width - ksize + 2*pad)/stride + 1;
    int num_threads = height * width * channels;
    col2im_gpu_kernel<<<(num_threads + blockSize -1 )/blockSize, blockSize>>>(data_col, channels, height, width,
                                                ksize, stride, pad, height_col, width_col, im);
}