#include "hip/hip_runtime.h"
#include "gemm.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define BLOCKSIZE 16

__global__ void gemm_nn_kernel( int m, int n, int k, int alpha, 
			 	                float* a, int la, 
			 	                float *b, int lb, 
			 	                float *c, int lc){

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int i;
    float sum = 0.;
    for(i=0; i<k; ++i){
        sum += alpha * a[row * la + k] * b[k * lb + col];
    }
    c[row * lc + col] = sum;
}


void gemm_gpu_nn(int m, int n, int k, int alpha, 
			 	 float* a, int la, 
			 	 float *b, int lb, 
			 	 float *c, int lc){
    float *d_a, *d_b, *d_c;
    size_t size_a = m * k * sizeof(float);
    size_t size_b = k * n * sizeof(float);
    size_t size_c = m * n * sizeof(float);

    hipMallocManaged((void**)&d_a, size_a);
    hipMallocManaged((void**)&d_b, size_b);
    hipMallocManaged((void**)&d_c, size_c);

    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
    dim3 gridSize((m + BLOCKSIZE - 1)/BLOCKSIZE, (n + BLOCKSIZE - 1)/BLOCKSIZE);

    gemm_nn_kernel<<<gridSize, blockSize>>>(m, n, k, alpha, d_a, la, d_b, lb, d_c, lc);
    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
    
}