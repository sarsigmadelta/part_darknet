#include "hip/hip_runtime.h"
#include "gemm.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define BLOCKSIZE 16

__global__ void gemm_nn_kernel( int m, int n, int k, float alpha, 
			 	                float* a, int la, 
			 	                float *b, int lb, 
			 	                float *c, int lc){

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int i;
    float sum = 0.;
    for(i=0; i<k; ++i){
        sum += alpha * a[row * la + i] * b[i * lb + col];
    }
    c[row * lc + col] = sum;
    //printf("%d\t%d\t%f\n", row, col, c[row * lc + col]);
}


void gemm_gpu_nn(int m, int n, int k, float alpha, 
			 	 float* a, int la, 
			 	 float *b, int lb, 
			 	 float *c, int lc){
    float *d_a, *d_b, *d_c;
    size_t size_a = m * k * sizeof(float);
    size_t size_b = k * n * sizeof(float);
    size_t size_c = m * n * sizeof(float);

    hipError_t cudaStatus1 = hipMallocManaged((void**)&d_a, size_a);
    hipError_t cudaStatus2 = hipMallocManaged((void**)&d_b, size_b);
    hipError_t cudaStatus3 = hipMallocManaged((void**)&d_c, size_c);

    hipError_t cudaStatus4 = hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipError_t cudaStatus5 = hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
    dim3 gridSize((m + BLOCKSIZE - 1)/BLOCKSIZE, (n + BLOCKSIZE - 1)/BLOCKSIZE);
    gemm_nn_kernel<<<gridSize, blockSize>>>(m, n, k, alpha, d_a, la, d_b, lb, d_c, lc);

    hipError_t cudaStatus6 = hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);  

    if(cudaStatus1 != hipSuccess || cudaStatus2 != hipSuccess || cudaStatus3 != hipSuccess ||
       cudaStatus4 != hipSuccess || cudaStatus5 != hipSuccess || cudaStatus6 != hipSuccess){
        printf("kernel function failed...\n");
    }
    
}