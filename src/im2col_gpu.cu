#include "hip/hip_runtime.h"
#include "im2col.h"


#define BLOCK_SIZE 256

__device__ float get_float_from_ptr_gpu(float *im, 
                                        int channels, int height, int width,
                                        int h, int w, int c){
    if ((h >= height) | (w >= width)){
        return 0.;
    }
    int index = c * height * width + h * width + w;
    return im[index];
}


__global__ void im2col_kernel(float *im, int channels, int height, int width, 
                              int ksize, int stride, int pad, int out_height, 
                              int out_width, float *data_col){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int w_out = index % out_width;
    int h_out = index / out_width % out_height;
    int channel_in = index / out_width / out_height;
    int channel_out = channel_in * ksize * ksize;
    int w_in = w_out * stride - pad;
    int h_in = h_out * stride - pad;
    
    float *data_col_out = data_col;
    data_col_out += channel_out * out_height * out_width + h_out * out_width + w_out;

    const float *im_read = im + channel_in * height * width + h_in * width + w_in;

    int i,j;
    for(i=0; i<ksize; ++i){
        for(j=0; j<ksize; ++j){
            int h = h_in + i;
            int w = w_in + j;
            float val = get_float_from_ptr_gpu(im, channels, height, width, h, w, channel_in);
            *data_col_out = val;
            data_col_out += out_height * out_width;
        }
    }
}


void im2col_gpu(float *im, 
                int channels, int height, int width,
                int ksize, int stride, int pad, float *data_col){

    int out_height = (height + 2*pad - ksize) / stride + 1;
    int out_width = (width + 2*pad - ksize) / stride + 1;
    int channels_cols = ksize * ksize * channels;
    int totoal_threads = out_height * out_width * channels;

    int num_inputs = channels * height * width ; 
    int num_outputs = channels * ksize * ksize * out_height * out_width;
    float *im_d;
    float *data_col_d;

    hipMallocManaged((void**)&im_d, num_inputs * sizeof(float));
    hipMallocManaged((void**)&data_col_d, num_outputs * sizeof(float));

    hipMemcpy(im_d, im, num_inputs * sizeof(float), hipMemcpyHostToDevice);

    im2col_kernel<<<(totoal_threads + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(im, channels, height, 
                                                    width, ksize, stride, pad, out_height, out_width, 
                                                    data_col_d);

    hipMemcpy(data_col, data_col_d, num_outputs * sizeof(float), hipMemcpyDeviceToHost);


}


void show_im2col_gpu_result(image im, int ksize, int stride, int pad){
    printf("before im2col....\n");
    //show_image(&im);
    
    int out_h = (im.height + 2*pad - ksize) / stride + 1;
    int out_w = (im.width + 2*pad - ksize) / stride + 1;
    int channels_cols = ksize * ksize * im.channels;
    float *data_col = (float*)calloc(channels_cols * out_h * out_w, sizeof(float));

    im2col(im.data, 
            im.channels, im.height, im.width,
            ksize, stride, pad, data_col);
    
    printf("after im2col....\n");

/*    int i, j;
    for(i=0; i<channels_cols; ++i){
        for(int j=0; j<out_h * out_w; ++j){
            int index = i * out_h * out_w + j;
            printf("%f ", data_col[index]);
        }
        printf("\n");
    }
*/

}