#include "hip/hip_runtime.h"
#include "im2col.h"


#define BLOCK_SIZE 256

__device__ float get_float_from_ptr_gpu(float *im, 
                                        int channels, int height, int width,
                                        int h, int w, int c){
    if ((h >= height) | (w >= width)){
        return 0.;
    }
    int index = c * height * width + h * width + w;
    return im[index];
}


__global__ void im2col_kernel(){
    
}

void im2col_gpu(float *im, 
                int channels, int height, int width,
                int ksize, int stride, int pad, float *data_col){

    int out_h = (height + 2*pad - ksize) / stride + 1;
    int out_w = (width + 2*pad - ksize) / stride + 1;
    int channels_cols = ksize * ksize * channels;
    int totoal_threads = out_h * out_w * channels_cols;
    

}