#include "hip/hip_runtime.h"
#include "matrix_mul_shared.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__device__ float get_element(mat m, int row, int col){
    return m.data[row * m.stride + col];
}

__device__ void set_element(mat m, int row, int col, float val){
    m.data[row * m.stride + col] = val;
}

__device__ mat get_sub_matrix(mat m, int row, int col){
    mat sub_m;
    sub_m.height = BLOCK_SIZE;
    sub_m.width = BLOCK_SIZE;
    sub_m.stride = m.stride;
    sub_m.data = &m.data[m.stride * row * BLOCK_SIZE + col * BLOCK_SIZE];
    return sub_m;
}

__global__ void matrix_mul_kernel(mat a, mat b, mat c){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int i;
    float cval = 0.;
    for(i=0; i<a.width; ++i){
        cval += a.data[row * a.width + i] * b.data[i * b.width + col];
    }
    set_element(c, row, col, cval);
}

__global__ void matrix_mul_s_kernel(mat a, mat b, mat c){
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    mat c_sub = get_sub_matrix(c, by, bx);

    float Cval = 0.;
    int m;
    for(m=0; m<a.height / BLOCK_SIZE; ++m){
        mat a_sub = get_sub_matrix(a, by, m);
        mat b_sub = get_sub_matrix(b, m, bx);

        __shared__ float sub_as[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float sub_bs[BLOCK_SIZE][BLOCK_SIZE];

        sub_as[ty][tx] = get_element(a_sub, ty, tx);
        sub_bs[ty][tx] = get_element(b_sub, ty, tx);
        
        __syncthreads();

        for(int e=0; e<BLOCK_SIZE; ++e){
            Cval += sub_as[ty][e] * sub_bs[e][tx];
        }

    }
    set_element(c_sub, ty, tx, Cval);
}

void matrix_mul_s_called(mat A, mat B, mat C){
    mat *d_A, *d_B, *d_C;

    size_t size_A = A.width * A.height * sizeof(float);
    size_t size_B = B.width * B.height * sizeof(float);
    size_t size_C = C.width * C.height * sizeof(float);

    hipMallocManaged((void**)&d_A, sizeof(mat));
    hipMallocManaged((void**)&d_B, sizeof(mat));
    hipMallocManaged((void**)&d_C, sizeof(mat));

    hipMallocManaged((void**)&d_A->data, size_A);
    hipMallocManaged((void**)&d_B->data, size_B);
    hipMallocManaged((void**)&d_C->data, size_C);

    d_A->height = A.height;
    d_A->width = A.width;
    d_A->stride = A.stride;

    d_B->height = B.height;
    d_B->width = B.width;
    d_B->stride = B.stride;

    d_C->height = C.height;
    d_C->width = C.width;
    d_C->stride = C.stride;

    hipMemcpy(d_A->data, A.data, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B->data, B.data, size_B, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize( (C.height + BLOCK_SIZE - 1) / BLOCK_SIZE,    (C.width + BLOCK_SIZE - 1) / BLOCK_SIZE);

    //matrix_mul_s_kernel<<<gridSize, blockSize>>>(*d_A, *d_B, *d_C);
    matrix_mul_kernel<<<gridSize, blockSize>>>(*d_A, *d_B, *d_C);

    hipMemcpy(C.data, d_C->data, size_C, hipMemcpyDeviceToHost);
    
}

void test_matrix_mul_s(){
    int height = 1024;
    int width = 1024;
    
    mat a = get_random_mat(height, width);
    mat b = get_random_mat(height, width);
    mat c = get_random_mat(height, width);

    matrix_mul_s_called(a, b, c);
    compare_result(a, b, c);

}
